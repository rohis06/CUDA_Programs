#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define N 1024  // Size of the matrix (N x N)
#define BLOCK_SIZE 32

__global__ void matrix_mul(float *C, float *A, float *B, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (row < n && col < n) {
        for (int i = 0; i < n; i++) {
            sum += A[row * n + i] * B[i * n + col];
        }

        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize host matrices
    for (int i = 0; i < N * N; i++) {
        A[i] = 1.0f;
        B[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Transfer data from host to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Setup the execution configuration
    int grid_dim = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_dim, grid_dim);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Executing kernel with a single thread
    matrix_mul<<<dimGrid, dimBlock>>>(d_C, d_A, d_B, N);

    // Transfer the result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N * N; i++) {
        if (C[i] != N * 2.0f) {
            printf("Error: element C[%d] = %f\n", i, C[i]);
            return -1;
        }
    }

    printf("PASSED\n");

    // Deallocate device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Deallocate host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
